
#include <hip/hip_runtime.h>
#include <stdio.h>


// Generic utils

typedef float3 pixel;

void check_result(hipError_t value) {
	hipError_t status = value;
	if (status != hipSuccess) {
		printf("Error %s at line %d in file %s\n",
			hipGetErrorString(status), __LINE__, __FILE__);
		// exit(1);
	}
}

__device__ float3 operator+(const float3 &a, const float3 &b) {
	return make_float3(a.x + b.x,a.y + b.y,a.z + b.z);
}

__device__ float3 operator*(const float3 &a, const float &b) {
	return make_float3(a.x * b, a.y * b, a.z * b);
}

__device__ float length(const float3 &vec) {
	return sqrt(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float3 normalize(const float3 vec) {
	float inverted_len = 1.0f / length(vec);
	return vec * inverted_len;
}


// Raymarcher

typedef struct {
	float3 o;
	float3 d;
} ray;

__device__ ray get_ray(const float& u, const float& v) {
	ray r;
	r.o = make_float3(-5.0, 0.0, 0.0);
	r.d = normalize(make_float3(1.0, u, v));
	return r;
}

__device__ float mandelbulb_de(float3 pos) {
	// pos = fmod(fabs(pos), 4.0) - 2.0;
	float3 z = pos;
	float dr = 1.0;
	float r = 0.0;
	int Iterations = 4;
	float Bailout = 4.0;
	float Power = 8.0;
	for(int i = 0; i < Iterations; i++) {
		r = length(z);
		if (r > Bailout) break;

		// convert to polar coordinates
		float theta = acos(z.z / r);
		float phi = atan2(z.y, z.x);
		dr = powf(r, Power - 1.0) * Power * dr + 1.0;

		// scale and rotate the point
		float zr = pow(r, Power);
		theta = theta * Power;
		phi = phi * Power;

		// convert back to cartesian coordinates
		z = make_float3(sin(theta) * cos(phi),
				sin(phi) * sin(theta), cos(theta)) * zr;
		z = z + pos;
		//z += pos * cos(time * 2.0);
	}
	return 0.5 * log(r) * r / dr;
}

__device__ float march(ray r) {
	float total_dist = 0.0;
	int max_ray_steps = 64;
	float min_distance = 0.002;

	int steps;
	for (steps = 0; steps < max_ray_steps; ++steps) {
		float3 p = r.o + r.d * total_dist;
		float distance = mandelbulb_de(p);
		total_dist += distance;
		if (distance < min_distance) break;
	}
	return 1.0 - (float) steps / (float) max_ray_steps;
}


// Main kernel

__global__ void d_main(
	pixel* screen_buffer,
	const size_t &width,
	const size_t &height
) {
	size_t x = (blockIdx.x * blockDim.x) + threadIdx.x;
	size_t y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if((x < width && y < height) && false) {
		float min_w_h = (float) min(width, height);

		float ar = (float) width / (float) height;
		float u = (float) x / min_w_h - ar * 0.5f;
		float v = (float) y / min_w_h - 0.5f;

		ray r = get_ray(u, v);
		float c = march(r) * 255.0f;
		float3 color = make_float3(c, c, c);
	}
	screen_buffer[y * width + x] = make_float3(255.0f, 255.0f, 255.0f);
}

int main(int argc, char** argv) {
	printf("Mandelbulb\n");
	
	size_t width = 32;
	size_t height = 32;
	size_t num_pixels = width * height;

	size_t group_width = 1;
	size_t group_height = 1;

	// Setup buffers
	pixel* h_screen_buff;
	pixel* d_screen_buff;
	check_result(hipHostMalloc(&h_screen_buff, num_pixels * sizeof(pixel), hipHostMallocDefault));
	check_result(hipMalloc(&d_screen_buff, num_pixels * sizeof(pixel)));

	dim3 block_dim(width / group_width, height / group_height);
	dim3 group_dim(group_width, group_height);

	// Execute on devicie
	printf("Starting kernel execution...\n");
	d_main<<<block_dim, group_dim>>>(d_screen_buff, width, height);
	printf("Kernel execution ended.\n");

	printf("Reading screan buffer from device...\n");
	check_result(hipMemcpy(h_screen_buff, d_screen_buff, num_pixels * sizeof(pixel), hipMemcpyDeviceToHost));
	printf("Done.\n");

	for(size_t y = 0;y < height;y++) {
		for(size_t x = 0;x < width;x++) {
			printf("%i ", (int) h_screen_buff[y * width + x].x);
		}
		printf("\n");
	}

	hipHostFree(h_screen_buff);
	hipFree(d_screen_buff);

	return 0;
}

